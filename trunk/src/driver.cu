#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <stdint.h>
#include <sys/time.h>

#include "Bitmap.h"
#include "Image.h"
#include "timer.h"
//#include "cuda_utils.h"

#define ALPHABET_SIZE 128
#define MAX_THREADS_PER_BLOCK 1024
#define min(a,b) (((a) < (b)) ? (a) : (b)) 

__global__ void init_precompute( uint8_t* precompute, uint8_t* pixel_row, int* prop );

int main(int argc, char* argv[])
{
	if(argc != 2)
	{
		printf("Usage : bin/exec filename\n");
		return 0;
	}
	char* filename = argv[1];
	Image I(filename);

	struct timeval t0, t1;
	double elapsed = 0.0;

	gettimeofday( &t0, NULL );	
	
	int image_width = I.WIDTH;
	int image_height = I.HEIGHT;
	int image_size = image_height * image_width;

	int NUM_THREADS = min( MAX_THREADS_PER_BLOCK , image_width );
	int BLOCK_CHUNK = image_width / MAX_THREADS_PER_BLOCK;
	int NUM_BLOCKS  = ALPHABET_SIZE * BLOCK_CHUNK;

	int prop[4];

	prop[0] = image_width / MAX_THREADS_PER_BLOCK;
	prop[1] = image_size;
	prop[2] = 0;
	prop[3] = MAX_THREADS_PER_BLOCK;

	// HOST POINTERS
	uint8_t* precompute = (uint8_t*) malloc( ALPHABET_SIZE * image_size * sizeof(uint8_t) );

	// DEVICE POINTERS
	uint8_t* dev_precompute;
	uint8_t* dev_pixel_row;

	
	
		// MAKE SPACE ON GPU
		
		hipMalloc( &dev_precompute, sizeof(uint8_t) * ALPHABET_SIZE * image_size  );
		hipMalloc( &dev_pixel_row, sizeof(uint8_t) * image_width );


		
		// INIT_PRECOMPUTE
		
		for( int i = 0; i < image_height; i++ ){

			prop[2] = i;

			uint8_t* pixel_row = (uint8_t*) malloc( sizeof(uint8_t) * image_width );

			for( int j = 0; j < image_width; j++ ){

				pixel_row[j] = I.p[ i * image_width + j ];

			}


			// COPY FROM CPU TO GPU

			hipMemcpy( &dev_pixel_row, pixel_row, sizeof( uint8_t ) * image_size, hipMemcpyHostToDevice );

			
			// KERNEL INVOCATION
			init_precompute<<<NUM_BLOCKS, NUM_THREADS>>>( precompute, pixel_row, prop );

			hipDeviceSynchronize();

	
			// COPY FROM GPU TO CPU

			hipMemcpy( &precompute[ i * image_size ], &dev_precompute, sizeof( uint8_t ) * ALPHABET_SIZE * image_size, hipMemcpyDeviceToHost );
	

		}


	
	gettimeofday( &t1, NULL );
	
	elapsed = ( t1.tv_sec - t0.tv_sec ) + ( ( t1.tv_usec - t0.tv_usec ) / 1e6 );

	printf("Timer: %lf \n", elapsed );

	
	return 0 ;	
}
