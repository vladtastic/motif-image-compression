
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdint.h>
//#include "cuda_utils.h"

#define ALPHABET_SIZE 		128
#define MAX_THREADS_PER_BLOCK	1024
#define min(a,b) (((a) < (b)) ? (a) : (b))

__global__ void init_precompute( uint8_t* precompute, uint8_t* pixel_row, int* prop )
{

	// pack BLOCK_CHUNK, image_size, current_row_num and MAX_THREADS_PER_BLOCK in an int* array

	int BLOCK_CHUNK = prop[0];
	int image_size = prop[1];
	int current_row_num = prop[2];
	int mtpb = prop[3];

	uint8_t pix_val = pixel_row[ ( blockIdx.x % BLOCK_CHUNK ) * blockDim.x + threadIdx.x ];

	int my_letter = blockIdx.x / BLOCK_CHUNK;
	

		precompute[ ( ( blockIdx.x / BLOCK_CHUNK ) * image_size + current_row_num ) + threadIdx.x + ( ( blockIdx.x % BLOCK_CHUNK ) * mtpb ) ] = 1;


	if( pix_val == (uint8_t) my_letter ){

		precompute[ ( ( blockIdx.x / BLOCK_CHUNK ) * image_size + current_row_num ) + threadIdx.x + ( ( blockIdx.x % BLOCK_CHUNK ) * mtpb ) ] = 0;

	}
}
	
